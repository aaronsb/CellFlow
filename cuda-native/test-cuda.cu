#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        std::cout << "No CUDA devices found!" << std::endl;
        return 1;
    }
    
    std::cout << "Found " << deviceCount << " CUDA device(s):" << std::endl;
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        std::cout << "\nDevice " << i << ": " << prop.name << std::endl;
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Total memory: " << (prop.totalGlobalMem / 1024 / 1024) << " MB" << std::endl;
        std::cout << "  Multiprocessors: " << prop.multiProcessorCount << std::endl;
        std::cout << "  Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    }
    
    return 0;
}